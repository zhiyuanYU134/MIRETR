#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "cuda_util.h"
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "ball_query.h"

// input: new_points(b, c, m) points(b, c, n)
// output: idx(b, m, nsample)
__global__ void ball_query_v1_kernel(int b,
                                     int c,
                                     int n,
                                     int m,
                                     float radius,
                                     int nsample,
                                     const float* __restrict__ new_points,
                                     const float* __restrict__ points,
                                     long* __restrict__ idx) {
  int batch_index = blockIdx.x;
  points += batch_index * c * n;
  new_points += batch_index * c * m;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float d2 = 0;
      for (int i = 0; i < c; ++i) {
        float delta = new_points[i * m + j] - points[i * n + k];
        d2 += delta * delta;
      }
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void ball_query_v1_kernel_launcher(int b,
                                   int c,
                                   int n,
                                   int m,
                                   float radius,
                                   int nsample,
                                   const float* new_points,
                                   const float* points,
                                   long* idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ball_query_v1_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, c, n, m, radius, nsample, new_points, points, idx);

  CUDA_CHECK_ERRORS();
}

// input: new_points(b, c, m) points(b, c, n)
// output: idx(b, m, nsample)
__global__ void ball_query_v2_kernel(int seed,
                                     hiprandState* rand_states,
                                     int b,
                                     int c,
                                     int n,
                                     int m,
                                     float radius,
                                     int nsample,
                                     const float* __restrict__ new_points,
                                     const float* __restrict__ points,
                                     long* __restrict__ idx) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState* local_state = rand_states + id;

  // TODO: optimize: hiprand_init is slow.
  hiprand_init(seed, id, 0, local_state);
  // // A potentially faster but less accurate version:
  // hiprand_init(seed + id * 1337, 0, 0, &rand_states[id]);

  int batch_index = blockIdx.x;
  points += batch_index * c * n;
  new_points += batch_index * c * m;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    for (int k = 0, cnt = 0; k < n; ++k) {
      float d2 = 0;
      for (int i = 0; i < c; ++i) {
        float new_point = new_points[m * i + j];
        float point = points[n * i + k];
        d2 += (new_point - point) * (new_point - point);
      }
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        } else if (cnt < nsample) {
          idx[j * nsample + cnt] = k;
        } else {
          unsigned int r = hiprand_uniform(local_state) * (cnt + 1);
          if (r < nsample) {
            idx[j * nsample + r] = k;
          }
        }
        ++cnt;
      }
    }
  }
}

void ball_query_v2_kernel_launcher(int seed,
                                   int b,
                                   int c,
                                   int n,
                                   int m,
                                   float radius,
                                   int nsample,
                                   const float* new_points,
                                   const float* points,
                                   long* idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int grid_dim = b;
  int block_dim = opt_n_threads(m);
  int num_threads = grid_dim * block_dim;

  hiprandState* rand_states;
  hipMalloc((void**)&rand_states, num_threads * sizeof(hiprandState));

  ball_query_v2_kernel<<<grid_dim, block_dim, 0, stream>>>(
      seed, rand_states, b, c, n, m, radius, nsample, new_points, points, idx);

  hipFree(rand_states);

  CUDA_CHECK_ERRORS();
}

